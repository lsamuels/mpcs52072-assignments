
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__ void mykernel(void) {
	printf("Hello");

}

int main(void) {
    mykernel<<<1,1>>>();
    hipDeviceSynchronize(); 
	printf(" World!\n");
	return 0;
}
